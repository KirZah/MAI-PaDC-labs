#include "hip/hip_runtime.h"

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

// if N >= 1024 then incorrect (cause of memory limits)
#define N 512  // 32 * x

//using namespace std;


// returns array of solutions
/*
float* find_equasion_solutions_gpu(function, border_left, width, pos_x) {
	// ���������� ����� ���������� �� ����������
	// (� ����� � ������� ������� �������� �������)
	float thread_x = border_left + (blockIdx.x + pos_x) * width;

	// count function value
	// (may be bettered by analysing the equasion or by using less float values)


	// compare resuslt with 0

}
//*/


float calculate_function_cpu(float a, float b, float x) {
	return a + b * x;
}

__device__ float calculate_function_gpu(float a, float b, float x) {
	return a + b * x;
}


/**
* FIXME: should return the amount of solutions found:
*    0 if solution is not found
*	-1 if error input
*	// -2 if found solution, but slightly outside the borders
*/
float* find_solution_borders_cpu(float* borders_ptr, float a, float b,
		float border_left, float border_right, float width, float pos_x) {
	///printf("\t==========================\n");
	///printf("\tfind_solution_borders_cpu:\n");
	///printf("\t--------------------------\n");
	int intervals_n = (int)((border_right - border_left) / width);
	///printf("\t[%f, %f] - borders\n", border_left, border_right);
	///printf("\twidth = %f\n", width);
	///printf("\tintervals_n = %d\n", intervals_n);
	if (intervals_n < 2) {
		printf("\tERROR: intervals_n = %d\n", intervals_n);
		system("pause");
		exit(1);
	}
	if ((border_right - border_left) < width * 2) {
		printf("\tERROR: incorrect width;  intervals_n = %d\n", intervals_n);
		system("pause");
		exit(1);
	}

	float f_x, f_x_, x, x_;

	bool solution_is_found = false;
	// ���������� ����� ���������� �� ���������� 
	// (� ����� � ������� ������� �������� ������� ��� ������ ����)
	int idx = 0;
	for (; idx < intervals_n+1; idx++) { // intervals_n + 1 is because: |"---"---"-|
		x = border_left + (idx + pos_x) * width;  // for cpu

		// count function value 
		// (may be bettered by analysing the equasion or by using less float values)
		f_x = calculate_function_cpu(a, b, x);
		if (idx == 0) {
			f_x_ = f_x;
		}

		// compare resuslt with 0 (finish cycle if soulution is found)
		if ((f_x_ > 0) && (f_x < 0) || (f_x_ < 0) && (f_x > 0)) {
			solution_is_found = true;
			///printf("\tFOUND solution in the %d interval\n", idx);
			break;
		}

		//remember if prev was less or more than 0
		x_ = x;
		f_x_ = f_x;
	}

	if (!solution_is_found) { // FIXME?
		///printf("\tSOLUTION IS NOT FOUND! checking if there's solution is beeween [x, right_border]\n");
		x = border_right;
		f_x = calculate_function_cpu(a, b, x);
		if ((f_x_ > 0) && (f_x < 0) || (f_x_ < 0) && (f_x > 0)) {
			///printf("\tSolution IS FOUND in the LAST GAP!\n");
		}
	}

	borders_ptr[0] = x_;
	borders_ptr[1] = x;

	///printf("\tf(x) in [%f, %f]\n", f_x_, f_x);
	///printf("\tx    in [%f, %f]\n", borders_ptr[0], borders_ptr[1]);
	//getchar();

	///printf("\t==========================\n\n");
	return borders_ptr;
}

float find_solution_cpu(float a, float b, float border_left,
		float border_right, float pos_x, float eps, int intervals_n) {
	printf("=======================\n");
	printf("count_solution_time_cpu\n");
	printf("-----------------------\n");

	// Dynamically allocate memory using malloc()
	float* borders_ptr = (float*)malloc(2 * sizeof(float));
	if (borders_ptr == NULL) {
		printf("ooooooops\n");
		getchar();
		exit(1);
	}
	borders_ptr[0] = border_left;
	borders_ptr[1] = border_right;


	float radius = border_right - border_left;
	float width = radius / intervals_n;

	printf("[%f, %f] - borders\n", borders_ptr[0], borders_ptr[1]);
	printf("radius = %f, \t", radius);
	printf("width  = %f\n", width);
	eps *= 2;
	printf("eps = %f\n", eps);

	int iterations = 0;
	while (radius > eps) {
		++iterations;
		borders_ptr = find_solution_borders_cpu(borders_ptr,
			a, b, borders_ptr[0], borders_ptr[1], width, pos_x);
		printf("RESULT %d: x in [%f, %f]\n", iterations, borders_ptr[0], borders_ptr[1]);

		radius = borders_ptr[1] - borders_ptr[0]; // abs() suppports only integer values!
		width = radius / intervals_n;

		///printf("radius = %f\t", radius);
		///printf("width  = %f\n\n", width);
		//system("pause");
	}
	printf("=======================\n");
	printf("FINAL RESULT: x in [%f, %f]\n", borders_ptr[0], borders_ptr[1]);
	float result = borders_ptr[0] + (borders_ptr[1] - borders_ptr[0]) / 2;
	eps /= 2;
	printf("x = %f +- %f\n", result, eps);
	printf("Iterations = %d\n\n", iterations);

	printf("radius = %f\n", radius);
	printf("eps    = %f\n", eps);
	printf("=======================\n\n");
	free(borders_ptr);
	return result;
}

double count_solution_time_cpu(float a, float b, float border_left,
		float border_right, float pos_x, float eps, int intervals_n) {
	clock_t begin = clock();
	float result = \
		find_solution_cpu(a, b, border_left, border_right, pos_x, eps, intervals_n);
	clock_t end = clock();
	return (double)(end - begin) / CLOCKS_PER_SEC;
}


// LINEAR PART IS ON GPU

__device__ void find_solution_borders_gpu_device(float* borders_ptr, float a, float b,
	float border_left, float border_right, float width, float pos_x) {
	/*
	printf("blockIdx.x = %d,\t threadIdx.x = %d\n", blockIdx.x, threadIdx.x);
	//float x = border_left + (blockIdx.x + pos_x) * width;  //for gpu
	//printf("res = %f\n", calculate_function_gpu(a, b, x));
	//*/

	printf("\t==========================\n");
	printf("\tfind_solution_borders_cpu:\n");
	printf("\t--------------------------\n");
	printf("\t[%f, %f] - borders\n", border_left, border_right);
	printf("\twidth = %f\n", width);
	int intervals_n = (int)((border_right - border_left) / width);
	printf("\tintervals_n = %d\n", intervals_n);
	if (intervals_n < 2) {
		printf("\tERROR: intervals_n = %d\n", intervals_n);
		//system("pause");
		//exit(1);
	}
	if ((border_right - border_left) < width * 2) {
		printf("\tERROR: incorrect width;  intervals_n = %d\n", intervals_n);
		//system("pause");
		//exit(1);
	}

	/////////////
	// FIXME: NEED TO CHANGE LAST GAP

	/// get prev: was less or more than 0?
	float x_ = border_left + ((threadIdx.x) + pos_x) * width,
		f_x_ = calculate_function_gpu(a, b, x_);


	// count function value 
	float x = border_left + ((threadIdx.x + 1) + pos_x) * width;  //for gpu
	// intervals_n + 1 is because: |"---"---"-|
	float f_x = calculate_function_gpu(a, b, x);

	// compare result with 0 (finish cycle if soulution is found)
	if ((f_x_ > 0) && (f_x < 0) || (f_x_ < 0) && (f_x > 0)) {
		//solution_is_found = true;
		printf("\tFOUND solution in the %d thread!\n", threadIdx.x);

		borders_ptr[0] = x_;
		borders_ptr[1] = x;
		printf("\tf(x) in [%f, %f]\n", f_x_, f_x);
		printf("\tx    in [%f, %f]\n", borders_ptr[0], borders_ptr[1]);
	}

	// FIXME (get rid of gap cheking by making search a bit wider)
	// Checking te GAP
	/*
	printf("\tChecking if there's solution beeween [x, right_border]\n");

	x = border_right;  //for gpu
	f_x = calculate_function_gpu(a, b, x);
		
	if ((f_x_ > 0) && (f_x < 0) || (f_x_ < 0) && (f_x > 0)) {
		printf("\tSolution IS FOUND in the LAST GAP!\n");
	}
	//*/


	//////////////
	
	printf("\t==========================\n\n");
	//return borders_ptr;
}


__global__ void find_solution_gpu_on_device(float a, float b,
	float border_left, float border_right,
	float pos_x, float eps, int intervals_n) {
	/*
	//__syncthreads();
	cudaDeviceSynchronize();
	//printf("blockIdx.x = %d\n", blockIdx.x);
	__shared__

	find_solution_borders_gpu(float* borders_ptr, float a, float b,
			float border_left, float border_right, float width, float pos_x)
	//*/
	printf("=======================\n");
	printf("find_solution_gpu_on_device\n");
	printf("-----------------------\n");

	// Dynamically allocate memory using malloc()
	float* borders_ptr = (float*)malloc(2 * sizeof(float));
	if (borders_ptr == NULL) {
		printf("ERROR: ooooooops\n");
		//getchar();
		//exit(1);
	}
	borders_ptr[0] = border_left;
	borders_ptr[1] = border_right;


	float radius = border_right - border_left;
	float width = radius / intervals_n;

	printf("[%f, %f] - borders\n", borders_ptr[0], borders_ptr[1]);
	printf("radius = %f, \t", radius);
	printf("width  = %f\n", width);
	eps *= 2;
	printf("eps = %f\n", eps);

	int iterations = 0;
	while (radius > eps) {
		++iterations;
		// <<<1, intervals_n>>>
		find_solution_borders_gpu_device(borders_ptr,
			a, b, borders_ptr[0], borders_ptr[1], width, pos_x);

		__syncthreads();

		printf("RESULT : x in [%f, %f]\n", borders_ptr[0], borders_ptr[1]);

		radius = borders_ptr[1] - borders_ptr[0]; // abs() suppports only integer values!
		width = radius / intervals_n;

		printf("radius = %f\t", radius);
		printf("width  = %f\n\n", width);
		//system("pause");
	}
	printf("=======================\n");
	printf("FINAL RESULT: x in [%f, %f]\n", borders_ptr[0], borders_ptr[1]);
	float result = borders_ptr[0] + (borders_ptr[1] - borders_ptr[0]) / 2;
	eps /= 2;
	printf("x = %f +- %f\n", result, eps);
	printf("Iterations = %d\n\n", iterations);

	printf("radius = %f\n", radius);
	printf("eps    = %f\n", eps);
	printf("=======================\n\n");
	free(borders_ptr);
}


double count_solution_time_gpu_device(float a, float b,
	float border_left, float border_right,
	float pos_x, float eps, int intervals_n) {
	clock_t begin = clock();

	int solutiouns_n = 1;	//AMOUNT OF ABLE SOLUTIONS
	float* result = new float[solutiouns_n];
	float* result_dev = NULL;
	hipMalloc((void**)&result_dev, solutiouns_n * sizeof(float));

	//dim3 threads = dim3(32, 1, 1);
	//dim3 blocks  = dim3(intervals_n / threads.x, 1, 1);



	// <<<1, 1>>>
	find_solution_gpu_on_device<<<1, intervals_n>>> (a, b,
		border_left, border_right,
		pos_x, eps, intervals_n
		);

	//system("pause");
	hipMemcpy(&result, result_dev, solutiouns_n * sizeof(float), hipMemcpyDeviceToHost);

	//system("pause");
	printf("--------------------------");
	printf("Result = %f\n", &result[0]);
	hipFree(result_dev);

	free(result);

	//cudaDeviceReset();
	clock_t end = clock();
	return (double)(end - begin) / CLOCKS_PER_SEC;
}


// LINEAR PART IS ON CPU

__global__ void find_solution_borders_gpu_global(float* borders_ptr_dev, float a, float b,
	float border_left, float border_right, float width, float pos_x) {
	/*
	printf("blockIdx.x = %d,\t threadIdx.x = %d\n", blockIdx.x, threadIdx.x);
	//float x = border_left + (blockIdx.x + pos_x) * width;  //for gpu
	//printf("res = %f\n", calculate_function_gpu(a, b, x));
	//*/

	int intervals_n = (int)((border_right - border_left) / width);
	if (intervals_n < 2) {
		printf("\tthreadIdx.x = %d: \tERROR: intervals_n = %d\n", threadIdx.x, intervals_n);
		//system("pause");
		//exit(1);
	}
	if ((border_right - border_left) < width * 2) {
		printf("\tthreadIdx.x = %d: \tERROR: incorrect width;  intervals_n = %d\n", threadIdx.x, intervals_n);
		//system("pause");
		//exit(1);
	}

	__syncthreads();
	/////////////
	// FIXME: NEED TO CHANGE LAST GAP

	/// get prev: was less or more than 0?
	float x_ = border_left + ((threadIdx.x) + pos_x) * width,
		f_x_ = calculate_function_gpu(a, b, x_);


	// count function value 
	float x = border_left + ((threadIdx.x + 1) + pos_x) * width;  //for gpu
	// intervals_n + 1 is because: |"---"---"-|
	float f_x = calculate_function_gpu(a, b, x);

	// compare result with 0 (finish cycle if soulution is found)
	if ((f_x_ > 0) && (f_x < 0) || (f_x_ < 0) && (f_x > 0)) {
		///printf("\tthreadIdx.x = %d: \t==========================\n", threadIdx.x);
		///printf("\tthreadIdx.x = %d: \t==========================\n", threadIdx.x);
		///printf("\tthreadIdx.x = %d: \tfind_solution_borders_cpu:\n", threadIdx.x);
		///printf("\tthreadIdx.x = %d: \t--------------------------\n", threadIdx.x);
		///printf("\tthreadIdx.x = %d: \t[%f, %f] - borders\n", threadIdx.x, border_left, border_right);
		///printf("\tthreadIdx.x = %d: \twidth = %f\n", threadIdx.x, width);
		///printf("\tthreadIdx.x = %d: \tintervals_n = %d\n", threadIdx.x, intervals_n);
		///printf("\tthreadIdx.x = %d: \tFOUND solution in the %d thread!\n", threadIdx.x, threadIdx.x);

		borders_ptr_dev[0] = x_;
		borders_ptr_dev[1] = x;
		///printf("\tthreadIdx.x = %d: \tf(x) in [%f, %f]\n", threadIdx.x, f_x_, f_x);
		///printf("\tthreadIdx.x = %d: \tx    in [%f, %f]\n", threadIdx.x, borders_ptr_dev[0], borders_ptr_dev[1]);
		///printf("\tthreadIdx.x = %d: \t==========================\n\n", threadIdx.x);
	}
	__syncthreads();

	// FIXME (get rid of gap cheking by making search a bit wider)
	// Checking te GAP
	/*
	printf("\tChecking if there's solution beeween [x, right_border]\n");

	x = border_right;  //for gpu
	f_x = calculate_function_gpu(a, b, x);

	if ((f_x_ > 0) && (f_x < 0) || (f_x_ < 0) && (f_x > 0)) {
		printf("\tSolution IS FOUND in the LAST GAP!\n");
	}
	//*/


	//////////////

	//return borders_ptr;
}



__host__ float find_solution_gpu_on_host(float a, float b,
		float border_left, float border_right,
		float pos_x, float eps, int intervals_n) {
	printf("=======================\n");
	printf("find_solution_gpu_on_host\n");
	printf("-----------------------\n");
	
	/*
	int solutiouns_n = 1;	//AMOUNT OF ABLE SOLUTIONS
	float* solutiouns = new float[solutiouns_n];
	float* solutiouns_dev = NULL;
	cudaMalloc((void**)&solutiouns_dev, solutiouns_n * sizeof(float));
	//dim3 threads = dim3(32, 1, 1);
	//dim3 blocks  = dim3(intervals_n / threads.x, 1, 1);
	float* solutiouns_dev = (float*)malloc(1 * sizeof(float));
	//*/

	int borders_n = 2 * 1; // borders (left and right) amount	-	2 * N 
	// Dynamically allocate memory using malloc() on CPU
	float* borders_ptr = (float*)malloc(borders_n * sizeof(float));
	// Dynamically allocate memory using malloc() on GPU
	float* borders_ptr_dev = NULL;

	// init before cycle
	borders_ptr[0] = border_left;
	borders_ptr[1] = border_right;
	float radius = borders_ptr[1] - borders_ptr[0];
	float width = radius / intervals_n;
	eps *= 2;
	printf("[%f, %f] - borders\n", borders_ptr[0], borders_ptr[1]);
	printf("radius = %f, \t", radius);
	printf("width  = %f\n", width);
	printf("eps = %f\n", eps);

	int iterations = 0;
	while (radius > eps) {
		++iterations;
		///printf("000000.\n");
		hipMalloc((void**)&borders_ptr_dev, borders_n * sizeof(float));
		find_solution_borders_gpu_global<<<1, intervals_n>>> (borders_ptr_dev,
				a, b, borders_ptr[0], borders_ptr[1], width, pos_x);
		//__syncthreads();
		///printf("111111.\n");
		//cudaDeviceSynchronize();
		hipMemcpy(borders_ptr, borders_ptr_dev, borders_n * sizeof(float), hipMemcpyDeviceToHost);
		hipDeviceReset();
		radius = borders_ptr[1] - borders_ptr[0]; // abs() suppports only integer values!
		width = radius / intervals_n;
		printf("RESULT %d: x in [%f, %f]\n", iterations, borders_ptr[0], borders_ptr[1]);
		///printf("radius = %f\t", radius);
		///printf("width  = %f\n\n", width);

		///printf("222222.\n");
		//system("pause");
	}
	printf("=======================\n");
	printf("FINAL RESULT: x in [%f, %f]\n", borders_ptr[0], borders_ptr[1]);
	float result = borders_ptr[0] + (borders_ptr[1] - borders_ptr[0]) / 2;
	eps /= 2;

	printf("x = %f +- %f\n", result, eps);
	printf("Iterations = %d\n", iterations);
	printf("radius = %f\n", radius);
	printf("eps    = %f\n", eps);
	printf("=======================\n\n");
	hipFree(borders_ptr_dev);
	free(borders_ptr);
	return result;
}



double count_solution_time_gpu_host(float a, float b,
		float border_left, float border_right,
		float pos_x, float eps, int intervals_n) {
	clock_t begin = clock();
	///float result = 
	find_solution_gpu_on_host(a, b,
		border_left, border_right, 
		pos_x, eps, intervals_n
		);

	///printf("Result = %f +- %f\n", result, eps);
	//cudaDeviceReset();
	clock_t end = clock();
	return (double)(end - begin) / CLOCKS_PER_SEC;
}


int func(void (*f)(int)) {
	return 1;
};

void do_smth(int a) { printf("%d\n", a); };

int main(void) {
	//read equasion
	//func(do_smth);
	//printf("%d\n");
	//parse equasion


	//rewrite equasion in needed format: f(x) = 0;


	//get_cpu_calculations_time


	//calculate_using_gpu_calculations_time


	//printf("\nPress 'Enter' to write numbers using CPU..\n");
	//getchar();


	// float pos_x = 0; // ����� ������ ���, ����� �� ������������ ������ �������� (����� ������� ����� � [border_left, border_left + pos_x])
	// FIXME: � ������ ������ ��� ��� ������� (x_ < border_right) & (x > border_right) ����� ��������� ���� �� ������� � [x, border_right]
	float a = 2., b = 3.,
		border_left = -10000., 
		border_right = 10000.,
		pos_x = 0, // ����� ������ ���, ����� �� ������������ ������ ��������
		eps = 0.1;
	int intervals_n = N;

	double cpu_time = count_solution_time_cpu(a, b, border_left, border_right, pos_x, eps, intervals_n);
	printf("\nPress 'Enter' to write numbers using GPU..\n");
	//system("pause");
	//getchar();
	double gpu_time = count_solution_time_gpu_host(a, b, border_left, border_right, pos_x, eps, intervals_n);
	printf("\n\n----------------------------------------------------\n");
	printf("Parameters:\n\tN = %d\n", N);
	printf("\n-------------\n");
	printf("Results:\n");
	printf("\tFinding the solution on CPU took %.2f seconds\n", cpu_time);
	printf("\tFinding the solution on GPU took %.2f seconds\n", gpu_time);
	printf("----------------------------------------------------\n\n");
	printf("Wow!!\n");
}




/*���� ������ � CUDA*/
/*
��. �����������
Shared ������� ������ ����� ��� ����������
constant
texture


|------------------------------------------------------|
| ������������	|	  ������	|	  ��� �������	   |
|  ����������	|				|					   |
|-------------------------------|----------------------|
| __device__	|	  device	|	 R				   |
| __constant__	|  host/device	|  R / W			   |
| __shared__	|	  block		| RW / __syncthreads() |
|------------------------------------------------------|
P.s. ��� ��� ��������� �� ����������.


� ���������� ������ ���� ���
������ � ��� ��� �����: tex1D, tex2D, tex1Dfetch

tex1Dfetch(tex, int)	  -	�������� (������ ��� ��������� ��������,)
tex1D(), tex2D(), tex3D() -	texcudaArray


���������� ��������� ���� �������� ������, �������� ����� ���������� ��������������� �������� �

gridDim
blockDim

*/